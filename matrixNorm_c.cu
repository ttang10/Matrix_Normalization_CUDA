/* Matrix normalization.
 * Compile with "nvcc matrixNorm_c.cu -o matrixNorm_c.out" 
 */

/* This is a cuda program normalizing matrix using 32*1 dimension blocks 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
  int seed = 0;  /* Random seed */
  char uid[32]; /*User name */

  /* Read command-line arguments */
  srand(time_seed());  /* Randomize */

  if (argc == 3) {
    seed = atoi(argv[2]);
    srand(seed);
    printf("Random seed = %i\n", seed);
  } 
  if (argc >= 2) {
    N = atoi(argv[1]);
    if (N < 1 || N > MAXN) {
      printf("N = %i is out of range.\n", N);
      exit(0);
    }
  }
  else {
    printf("Usage: %s <matrix_dimension> [random seed]\n",
           argv[0]);    
    exit(0);
  }

  /* Print parameters */
  printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
  for (col = 0; col < N; col++) {
    for (row = 0; row < N; row++) {
      A[row][col] = (float)rand() / 32768.0;
      B[row][col] = 0.0;
    }
  }

}

/* Print input matrices */
void print_inputs() {
  int row, col;

  if (N < 10) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
  }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

int main(int argc, char **argv) {
  /* Timing variables */
  struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
  struct timezone tzdummy;
  clock_t etstart2, etstop2;  /* Elapsed times using times() */
  unsigned long long usecstart, usecstop;
  struct tms cputstart, cputstop;  /* CPU times for my processes */

  /* Process program parameters */
  parameters(argc, argv);

  /* Initialize A and B */
  initialize_inputs();

  /* Print input matrices */
  print_inputs();

  /* Start Clock */
  printf("\nStarting clock.\n");
  gettimeofday(&etstart, &tzdummy);
  etstart2 = times(&cputstart);

  /* Gaussian Elimination */
  matrixNorm();

  /* Stop Clock */
  gettimeofday(&etstop, &tzdummy);
  etstop2 = times(&cputstop);
  printf("Stopped clock.\n");
  usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
  usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

  /* Display output */
  print_B();

  /* Display timing results */
  printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

  printf("(CPU times are accurate to the nearest %g ms)\n",
	 1.0/(float)CLOCKS_PER_SEC * 1000.0);
  printf("My total CPU time for parent = %g ms.\n",
	 (float)( (cputstop.tms_utime + cputstop.tms_stime) -
		  (cputstart.tms_utime + cputstart.tms_stime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My system CPU time for parent = %g ms.\n",
	 (float)(cputstop.tms_stime - cputstart.tms_stime) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My total CPU time for child processes = %g ms.\n",
	 (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
		  (cputstart.tms_cutime + cputstart.tms_cstime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
      /* Contrary to the man pages, this appears not to include the parent */
  printf("--------------------------------------------\n");
  
  exit(0);
}

/* ------------------ Above Was Provided --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
 * defined in the beginning of this code.  B[][] is initialized to zeros.
 */

/* Kernel function */

__global__ void colCal(float* d_A, float* d_B, float* d_mu, float* d_sigma, int d_N, int d_mn) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col > d_N) return;
	for (int row = 0; row < d_N; row++) 
		d_mu[col] += d_A[row*d_mn+col];
	d_mu[col] /= (float) d_N;
	__syncthreads();
	for (int row = 0; row < d_N; row++) 
		d_sigma[col] += powf(d_A[row*d_mn+col] - d_mu[col], 2.0);
	d_sigma[col] /= (float) d_N;
	__syncthreads();
	for (int row=0; row < d_N; row++) {
		if (d_sigma[col] == 0.0) 
			d_B[row*d_mn+col] = 0.0;
		else
			d_B[row*d_mn+col] = (d_A[row*d_mn+col] - d_mu[col]) / d_sigma[col];
	}
}


void matrixNorm() {
  
  /* Define variables */
  float* d_A;
  float* d_B;
  int d_N = N;
  int d_mn = MAXN;
  float mu[N];
  memset(mu, 0.0, N*sizeof(float));
  float sigma[N];
  memset(sigma, 0.0, N*sizeof(float));
  float* d_mu;
  float* d_sigma;

  printf("Computing Parallel via CUDA.\n");
  
  /* Malloc space for device variables */

  hipMalloc((float**)&d_A, sizeof(float)*pow(MAXN, 2));
  hipMalloc((float**)&d_B, sizeof(float)*pow(MAXN, 2));
  hipMalloc((float**)&d_mu, sizeof(float)*N);
  hipMalloc((float**)&d_sigma, sizeof(float)*N);
  
  /* Copy the memory from host to device */

  hipMemcpy(d_A, (float*)A, sizeof(float)*pow(MAXN, 2), hipMemcpyHostToDevice);
  hipMemcpy(d_B, (float*)B, sizeof(float)*pow(MAXN, 2), hipMemcpyHostToDevice);
  hipMemcpy(d_mu, (float*)mu, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_sigma, (float*)sigma, sizeof(float)*N, hipMemcpyHostToDevice);
  
  /* define GPU grids and blocks */

  dim3 dimBlock(32, 1, 1);
  dim3 dimGrid(ceil(N/32), 1, 1);
  colCal<<<dimGrid, dimBlock>>>(d_A, d_B, d_mu, d_sigma, d_N, d_mn);
  
  /* Copy the value of B matrix back */

  hipMemcpy((float*)B, d_B, sizeof(float)*pow(MAXN, 2), hipMemcpyDeviceToHost);
  
  /* Free memory of device variables */
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_mu);
  hipFree(d_sigma);
}

